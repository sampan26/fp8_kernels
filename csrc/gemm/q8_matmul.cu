#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <torch/python.h>
#include "q8_gemm_api.cuh"

#include "cutlass/cutlass.h"
#include "cutlass/layout/layout.h"
#include <cute/tensor.hpp>

using namespace cute;

void matmul_fn(int8_t *A, int8_t *B, void *C, float* A_scale, float* B_scale, int B_A, int B_B, int M, int N, int K, bool fuse_gelu) {
    int Bs;
    TORCH_CHECK(B_A == B_B || (B_A == 1 || B_B == 1), "Batch size mismatch");

    if (B_A == 1 || B_B == 1) {
        Bs = B_A * B_B;
    }
    else if (B_A == B_B) {
        Bs = B_A;
    }
    auto bM = Int<128>{};
    auto bN = Int<128>{};
    auto bK = Int<64>{};
    auto bP = Int<2>{};

    using SmemLayoutAtom = decltype(composition(
        Swizzle<2, 4, 3>{},
        cute::make_layout(cute::make_shape(Int<8>{}, Int<bK>{}),
                    cute::make_stride(Int<bK>{}, Int<1>{}))));

    using SmemLayoutA = decltype(
        cute::tile_to_shape(SmemLayoutAtom{}, cute::make_shape(bM, bK, bP))
    );
    
    using SmemLayoutB = decltype(
        cute::tile_to_shape(SmemLayoutAtom{}, cute::make_shape(bN, bK, bP))
    );

    using mma_op = SM80_16x8x32_S32S8S8S32_TN;
    using mma_traits = MMA_Traits<mma_op>;
    using mma_atom = MMA_Atom<mma_traits>;

    static constexpr int WARP_ROWS = 2;
    static constexpr int WARP_COLS = 2;
    
    using mma_atom_shape = mma_traits::Shape_MNK;

    static constexpr int MMA_WARP_M = WARP_ROWS * get<0>(mma_atom_shape{});
    static constexpr int MMA_WARP_N = 1 * WARP_COLS * get<1>(mma_atom_shape{});
    static constexpr int MMA_WARP_K = 1 * get<2>(mma_atom_shape{});

    using MMA_WARP_Tile = decltype(
        make_layout(make_shape(Int<WARP_ROWS>{}, Int<WARP_COLS>{}, Int<1>{}))
    );
    using MMA_PARTITION_TILE = Tile<Int<MMA_WARP_M>, Int<MMA_WARP_N>, Int<MMA_WARP_K>>;
    using MMA = decltype(make_tiled_mma(mma_atom{}, MMA_WARP_Tile{}, MMA_PARTITION_TILE{}));

    using g2s_copy_op = SM80_CP_ASYNC_CACHEGLOBAL<cute::uint128_t>;
    using g2s_copy_traits = Copy_Traits<g2s_copy_op>;
    using g2s_copy_atom = Copy_Atom<g2s_copy_atom>;
    using G2SCopyA = decltype(
        make_tiled_copy(g2s_copy_atom{}, 
        make_layout(make_shape(Int<32>{}, Int<4>{}), make_stride(Int<4>{}, Int<1>{})),
        make_layout(make_shape(Int<1>{}, Int<16>{}))));
    using G2SCopyB = G2SCopyA;

    using s2r_copy_op_a = SM75_U32x4_LDSM_N;
    using s2r_copy_traits_a = Copy_Traits<s2r_copy_op_a>;
    using s2r_copy_atom_a = Copy_Atom<s2r_copy_traits_a, int8_t>;

    using s2r_copy_op_b = SM75_U32x2_LDSM_N;
    using s2r_copy_traits_b = Copy_Traits<s2r_copy_op_b>;
    using s2r_copy_atom_b = Copy_Atom<s2r_copy_traits_b, int8_t>;

    using SmemLayoutC = decltype(
        compostiion(
            Swizzle<2, 4, 3>{},
            make_layout(make_shape(Int<MMA_WARP_M>{}, Int<MMA_WARP_N*Int<4>{}>{})),
            make_stride(Int<MMA_WARP_N*Int<4>{}>{}, Int<1>{}))
    );

    using R2SCopyAtomC = Copy_Atom<UniversalCopy<cute::uint16_t>, float_e4m3_t>;
    using S2RCopyAtomC = Copy_Atom<UniversalCopy<cute::uint128_t>, float_e4m3_t>;

    using S2GCopyC =
        decltype(make_tiled_copy(S2GCopyAtomC{},
                                make_layout(make_shape(Int<32>{}, Int<4>{}),
                                            make_stride(Int<4>{}, Int<1>{})),
                                make_layout(make_shape(Int<1>{}, Int<16>{}))));

    int bX = (N + bN - 1) / bN;
    int bY = (M + bM - 1) / bM;
    int bZ = Bs;

    dim3 block(size(MMA{}));
    dim3 grid(bX, bY, bZ);

    static constexpr int shm_size_AB = cute::cosize(SmemLayoutA) + cute::cosize(SmemLayoutB);
    static constexpr int shm_size_C = cute::cosize(SmemLayoutC);
    static constexpr int kShmSize = cute::max(shm_size_AB, shm_size_C) * sizeof(cute::float_e4m3_t);

    int shm_size = kShmSize;
}



torch::Tensor q8_mm(torch::Tensor a, torch::Tensor a_scale, torch::Tensor b, torch::Tensor b_scale, bool fuse_gelu) {
    CHECK_INPUT(a);
    CHECK_INPUT(b);
    
    int m, n, k;

    int a_ndim = a.sizes().size();
    int b_ndim = b.sizes().size();

    int bs_a;
    if (a_ndim == 3) {
        bs_a = a.size(0);
        m = a.size(1);
    }
    else {
        bs_a = 1;
        m = a.size(0);
    }

    int bs_b;
    if (b_ndim == 3) {
        bs_b = b.size(0);
        n = b.size(1);
    }
    else {
        bs_b = 1;
        n = b.size(0);
    }

    k = a.size(a_ndim - 1);

    TORCH_CHECK(bs_a == bs_b || bs_a == 1 || bs_b == 1, "Batch missmatch");

    int B;
    if (a_ndim == 1 || b_ndim == 1) {
        B = bs_a * bs_b;
    }
    else {
        B = bs_a;
    }
    auto opts = a.options();
    auto out = torch::empty({B, m, n}, opts.dtype(torch::kFloat8_e4m3fn));

    matmul_fn(
        a.data_ptr<int8_t>(), 
        b.data_ptr<int8_t>(), 
        out.data_ptr(), 
        a_scale.data_ptr<float>(), 
        b_scale.data_ptr<float>(), 
        bs_a, bs_b, m, n, k, fuse_gelu
    );
    hipDeviceSynchronize();
    CUDA_ERROR_CHECK(hipGetLastError());

    return out;
}